#include "hip/hip_runtime.h"
/*************************************************************************
	
	Copyright (C) 2017	Evandro Taquary,
	 					Mateus Freitas, 
	 					Wellington Martins
	
	This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
	
*************************************************************************/

#include <cudutils.h>

__global__ void modcpy(void *destination, void *source, size_t destination_size, size_t source_size){

	int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	int pos;
 
	int ds = destination_size/sizeof(int4), ss = source_size/sizeof(int4);
	for(int i = idx; i < ds; i += gridDim.x * blockDim.x){
		pos = i % ss;
		reinterpret_cast<int4*>(destination)[i] = reinterpret_cast<int4*>(source)[pos];  
	}
}

__host__ __device__ int row_index( int i, int M ){
	M--;
    float m = M;
    float row = (-2*m - 1 + sqrt( (4*m*(m+1) - 8*(float)i - 7) )) / -2;
    if( row == (float)(int) row ) row -= 1;
    return (int) row;
}

__host__ __device__ int column_index( int i, int M ){
    int row = row_index( i, M);
    M--;
    return 1 + (i - M * row + row*(row+1) / 2);
}
