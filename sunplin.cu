#include "hip/hip_runtime.h"
/*************************************************************************
	
	Copyright (C) 2016	Evandro Taquary, Thiago Santos
	
	This program is free software: you can redistribute it and/or modify s
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
	
*************************************************************************/

#include <iostream>
#include <string>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "modcpy.h"
#include <regex>
#include <unordered_map>
#include <iomanip>
#include <sys/time.h>

using namespace std;

#define START_TIMER() \
		{ \
			gettimeofday(&tv, NULL); \
			start_time = tv.tv_sec * 1000000 + tv.tv_usec; \
		}
//return time measurement in s
#define STOP_TIMER(time_spent) \
		{ \
			gettimeofday(&tv, NULL); \
			end_time = tv.tv_sec * 1000000 + tv.tv_usec; \
			time_spent = ((double)(end_time-start_time))/1000000; \
		}

#define CHECK(call) \
		{ \
			const hipError_t error = call; \
			if (error != hipSuccess) { \
				cout << "Error: " << __FILE__ ": " << __LINE__ << ", "; \
				cout << "code: "<< error << ", reason: " << hipGetErrorString(error) << endl; \
				exit(EXIT_FAILURE); \
			} \
		}

#define FERR(file) \
		{ \
			if(!file.good()){ \
				cout << "Something went wrong while reading the file! Please try again." << endl; \
				cout << "Error: " << __FILE__ ": " << __LINE__ << ", " << endl; \
				exit(EXIT_FAILURE); \
			} \
		}

typedef struct {
	ushort side	: 1;
	ushort idx	: 15;
} paren_t;

#define NOCHILD USHRT_MAX		//16 bits
#define NOPARENT USHRT_MAX/2	//15 bits

class SoaTree {
	private:
		paren_t	*parent;	// nodes' parents or the subtrees' roots' indices where new nodes shall be inserted (MDCC)
		ushort	*lChild;	// nodes' left children
		ushort	*rChild;	// nodes' right children
		float	*branch;	// lengths of the nodes' branches (distance to the parent)
		float	*dRoot;		// distances between nodes and root (sum of the paths' branches)
		ushort	*inseq;		// vector with the sequence of indices of puts to be inserted
	public:
		__host__ SoaTree() = default;
		__host__ SoaTree(int num_nodes, int num_ins) {soalloc(num_nodes,num_ins);}
		__host__ SoaTree(int num_nodes, void* base) {setOffs(num_nodes,base);}
		__host__ __device__ void* getPtr() const {return (void*) parent;}
		__host__ __device__ void setOffs(int num_nodes, void* base);	//set pointers' offsets starting on base accordingly to data structure, # nodes and # insertions
		__host__ void setOffs(int num_nodes) {setOffs(num_nodes, parent);}  //set pointers' offsets starting on the first array accordingly to data structure, # nodes and # insertions
		__host__ static size_t getSize(int num_nodes, int num_ins)
		{
			size_t size = (3*sizeof(ushort) + 2*sizeof(float))*num_nodes + sizeof(ushort)*num_ins; //minimal amount of bytes needed to represent the tree 
			int r = size%sizeof(int4);
			size += r ? sizeof(int4)-r : 0;	//size of the tree padded to a multiple of sizeof(int4) (due to a GPU memory aligment requisite)
			return size;
		}
		__host__ void soalloc(int num_nodes, int num_ins)
		{
			void *ptr = malloc(getSize(num_nodes, num_ins));
			memset(ptr,0,getSize(num_nodes, num_ins));
			setOffs(num_nodes, ptr);
		}

		__host__ __device__ ushort	getParent	(int i) const {return parent[i].idx;}
		__host__ __device__ ushort	getSide		(int i) const {return parent[i].side;}
		__host__ __device__ ushort	getlChild	(int i) const {return lChild[i];}
		__host__ __device__ ushort	getrChild	(int i) const {return rChild[i];}
		__host__ __device__ ushort 	getInseq	(int i) const {return inseq[i];}
		__host__ __device__ float 	getBranch	(int i) const {return branch[i];}
		__host__ __device__ float	getdRoot	(int i) const {return dRoot[i];}

		__host__ __device__ void setParent	(ushort	val, int i)	{parent[i].idx	= val;}
		__host__ __device__ void setSide	(ushort	val, int i)	{parent[i].side	= val;}
		__host__ __device__ void setlChild	(ushort	val, int i)	{lChild[i]		= val;}
		__host__ __device__ void setrChild	(ushort	val, int i)	{rChild[i]		= val;}
		__host__ __device__ void setBranch	(float	val, int i)	{branch[i]		= val;}
		__host__ __device__ void setdRoot	(float	val, int i)	{dRoot[i]		= val;}
		__host__ __device__ void setInseq	(ushort	val, int i)	{inseq[i]		= val;}
};

void SoaTree::setOffs(int num_nodes, void* base) {
	parent 	= (paren_t*) base;
	lChild 	= (ushort*)	(parent+num_nodes);
	rChild 	= lChild	+ num_nodes;	
	branch 	=(float*)	(rChild+num_nodes);
	dRoot 	= branch	+ num_nodes;
	inseq 	=(ushort*)	(dRoot+num_nodes);
}

class HTree;

class DTree{

	protected:
		void *base;
		ushort nNodes;		// quantity of nodes on the tree(s) (including inserting species)
		ushort nInsSpc;		// quantity of absent species to be inserted
		ushort idxInsSpc;	// starting index for insertion of new species
		ushort idxInsAnc;	// starting index for insertion of new ancestors
		uint nTrees;		// quantity of trees holded by devData (default=1)
		size_t treeSize;	// size of one tree padded to multiple of sizeof(int4) (due to a GPU memory aligment requisite)
		SoaTree devData;	// struct of arrays to hold trees' data
	public:

		__host__ bool compareTo(HTree *h_tree);
		__host__ DTree() = default;
		__host__ DTree(int nNodes, int nInsSpc, int idxInsSpc, int idxInsAnc, int nTrees, size_t treeSize, void* ptr):
						nNodes(nNodes),
						nInsSpc(nInsSpc),
						idxInsSpc(idxInsSpc), 
						idxInsAnc(idxInsAnc),
						nTrees(nTrees),
						treeSize(treeSize){base=ptr; devData.setOffs(nNodes,ptr);}
		
		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */
		__host__ __device__ ushort	getnNodes	()		const {return nNodes;}
		__host__ __device__ ushort	getnInsSpc	()		const {return nInsSpc;}
		__host__ __device__ ushort	getIdxInsSpc()		const {return idxInsSpc;}
		__host__ __device__ ushort	getIdxInsAnc()		const {return idxInsAnc;}
		__host__ __device__ uint	getnTrees	()		const {return nTrees;}
		__host__ __device__ size_t	getSize		()		const {return treeSize;};

		__host__ __device__ ushort	getParent	(int i)	const {return devData.getParent(i);}
		__host__ __device__ ushort	getSide		(int i)	const {return devData.getSide(i);}
		__host__ __device__ ushort	getlChild	(int i)	const {return devData.getlChild(i);}
		__host__ __device__ ushort	getrChild	(int i)	const {return devData.getrChild(i);}
		__host__ __device__ float	getBranch	(int i)	const {return devData.getBranch(i);}
		__host__ __device__	float	getdRoot	(int i) const {return devData.getdRoot(i);}
		__host__ __device__ ushort	getInseq	(int i)	const {return devData.getInseq(i);}

		__device__ void	setTreeIdx(int i){devData.setOffs(nNodes,base+treeSize*i);}

		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */
		__device__ void	setParent	(ushort	val, int i)	{devData.setParent(val,i);}
		__device__ void	setSide		(ushort	val, int i)	{devData.setSide(val,i);}
		__device__ void	setlChild	(ushort	val, int i)	{devData.setlChild(val,i);}
		__device__ void	setrChild	(ushort	val, int i)	{devData.setrChild(val,i);}
		__device__ void	setBranch	(float	val, int i)	{devData.setBranch(val,i);}
		__device__ void	setdRoot	(float	val, int i)	{devData.setdRoot(val,i);}
		__device__ void	setInseq	(ushort	val, int i)	{devData.setInseq(val,i);}
		
		/* copy from GPU all the trees holded by the object and print them on the standard output  */
		__host__ void print(unordered_map<int,string> names);
		__host__ void print(unordered_map<int,string> names, int i);
		__host__ void free(){CHECK(hipFree(devData.getPtr()))}
};

void DTree::print(unordered_map<int,string> names){

	size_t rep_size = treeSize*nTrees;	
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, base, rep_size, hipMemcpyDeviceToHost));	
	SoaTree ht;
	string aux;
	int i,j;	
	cout.precision(4);
	cout.setf(ios::fixed, ios::floatfield);	
	cout << endl;
	for(i=0; i<nTrees; i++){
		cout<<"tree #"<<i<<endl;
		ht.setOffs(nNodes, h_replics+(treeSize*i));		
		for(j=0; j<nNodes; j++){
			aux = names[j]+"("+to_string(j)+")";
			cout << left << setw (40) << aux;
		} 
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getParent(j)!=NOPARENT ? names[ht.getParent(j)]+"("+to_string(ht.getParent(j))+")" : "-1";
			cout << left << setw (40) << aux;
		} 
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getSide(j)==1 ? "left" : "right";
			cout << left << setw (40) << aux;
		} 
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getlChild(j)!=NOCHILD ? names[ht.getlChild(j)]+"("+to_string(ht.getlChild(j))+")" : "-2";
			cout << left << setw (40) << aux;	
		}
		cout << endl;
		for(j=0; j<nNodes; j++) {
			aux = ht.getrChild(j)!=NOCHILD ? names[ht.getrChild(j)]+"("+to_string(ht.getrChild(j))+")" : "-2";
			cout << left << setw (40) << aux;
		}
		cout << endl;
		for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getBranch(j); cout << endl;
		for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getdRoot(j); cout << endl;
		for(j=0; j<nInsSpc; j++)cout << names[ht.getInseq(j)] << "("<< ht.getInseq(j) <<") ";
		cout << endl << endl;
	}
}

void DTree::print(unordered_map<int,string> names, int i){

	size_t rep_size = treeSize*nTrees;
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, devData.getPtr(), rep_size, hipMemcpyDeviceToHost));
	SoaTree ht;
	string aux;
	int j;
	cout.precision(4);
	cout.setf(ios::fixed, ios::floatfield);
	cout << endl;
	cout<<"tree #"<<i<<endl;
	ht.setOffs(nNodes, h_replics+(treeSize*i));
	for(j=0; j<nNodes; j++){
		aux = names[j]+"("+to_string(j)+")";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getParent(j)!=NOPARENT ? names[ht.getParent(j)]+"("+to_string(ht.getParent(j))+")" : "-1";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getSide(j)==1 ? "left" : "right";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getlChild(j)!=NOCHILD ? names[ht.getlChild(j)]+"("+to_string(ht.getlChild(j))+")" : "-2";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) {
		aux = ht.getrChild(j)!=NOCHILD ? names[ht.getrChild(j)]+"("+to_string(ht.getrChild(j))+")" : "-2";
		cout << left << setw (40) << aux;
	}
	cout << endl;
	for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getBranch(j); cout << endl;
	for(j=0; j<nNodes; j++) cout << left << setw (40) << ht.getdRoot(j); cout << endl;
	for(j=0; j<nInsSpc; j++)cout << names[ht.getInseq(j)] << "("<< ht.getInseq(j) <<") ";
	cout << endl << endl;

}

class HTree: public DTree{	
	private:
		SoaTree hostData;					// struct of arrays to hold the trees' data 		
		unordered_map<int, string> name;	// names of taxa fetched from newickf and PUT file
		ifstream newickf;					// stream object to manage input newick file
		ifstream putf;						// stream object to manage input PUT file
		int devId;							// id of the GPU where lies the tree 
		__host__ void setParams(string &fileLine, vector<string> &filePut);
		__host__ void parseTree(string fileLine, vector<string> filePut);
	public:
		__host__ HTree() = default;
		__host__ HTree(int dev_id=0, string nw_fname = "newick.tree", string pt_fname="put.list");
		__host__ DTree& gpuRep(int num_reps) const;
		
		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */		
		__host__ void setParent (int 	val, int i)	{hostData.setParent(val,i);}
		__host__ void setSide	(int 	val, int i)	{hostData.setSide(val,i);}
		__host__ void setlChild (int 	val, int i)	{hostData.setlChild(val,i);}
		__host__ void setrChild (int 	val, int i)	{hostData.setrChild(val,i);}
		__host__ void setBranch (float 	val, int i)	{hostData.setBranch(val,i);}
		__host__ void setdRoot  (float 	val, int i) {hostData.setdRoot(val,i);}
		__host__ void setInseq 	(int 	val, int i) {hostData.setInseq(val,i);}
		__host__ void setName 	(string val, int i) {name[i]=val;}
		
		/* TODO: THROW OVER/UNDERFLOW EXCEPTION */
		__host__ ushort	getParent	(int i) const	{return hostData.getParent(i);}
		__host__ ushort	getSide		(int i) const	{return hostData.getSide(i);}
		__host__ ushort	getlChild	(int i) const	{return hostData.getlChild(i);}
		__host__ ushort	getrChild	(int i) const 	{return hostData.getrChild(i);}
		__host__ float	getBranch	(int i) const 	{return hostData.getBranch(i);}
		__host__ float	getdRoot	(int i) const 	{return hostData.getdRoot(i);}
		__host__ ushort	getInseq	(int i) const 	{return hostData.getInseq(i);}
		__host__ string	getName		(int i)			{return name[i];}
		__host__ unordered_map<int, string> getNames(){return name;}
};

HTree::HTree(int dev_id, string nw_fname, string pt_fname){	
	long long start_time, end_time;
	struct timeval tv;

	void * d_tree;
	double time_spent;
	devId = dev_id;
	nTrees=1;
	CHECK(hipSetDevice(devId));

	START_TIMER();
	newickf.open(nw_fname);
	FERR(newickf);
	putf.open(pt_fname);
	FERR(putf);	
	string fileLine;
	vector<string> filePut;
	setParams(fileLine,filePut);	
	hostData.soalloc(nNodes,nInsSpc);
	treeSize = hostData.getSize(nNodes,nInsSpc);
	parseTree(fileLine,filePut);
	newickf.close();
	putf.close();
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to parse the files: "<<time_spent<<"s\n";	

	//make a copy of the tree on device side
	START_TIMER();
	CHECK(hipMalloc(&d_tree, treeSize));
	CHECK(hipMemcpy(d_tree, hostData.getPtr(), treeSize, hipMemcpyHostToDevice));	
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to copy backbone tree to GPU: "<<time_spent<<"s\n";
	base=d_tree;
	devData.setOffs(nNodes, d_tree);
}

bool DTree::compareTo(HTree *h_tree){
	if(treeSize != h_tree->getSize() || idxInsSpc != h_tree->getIdxInsSpc() || idxInsAnc != h_tree->getIdxInsAnc())
		return false;
	SoaTree tree;
	size_t rep_size = treeSize * nTrees;	
	void* h_replics = malloc(rep_size);
	CHECK(hipMemcpy(h_replics, devData.getPtr(), rep_size, hipMemcpyDeviceToHost));
	cout.precision(3);
	cout.setf(ios::fixed, ios::floatfield);	
	for(int j=0; j<nTrees; j++){
		tree.setOffs(nNodes, h_replics+treeSize*j);
		for(int i=0; i<nNodes; i++){
			if(	tree.getdRoot(i)	!= h_tree->getdRoot(i)	||
				tree.getBranch(i)	!= h_tree->getBranch(i) ||
				tree.getParent(i)	!= h_tree->getParent(i)	||
				tree.getSide(i)		!= h_tree->getSide(i)	||
				tree.getlChild(i)	!= h_tree->getlChild(i)	||
				tree.getrChild(i)	!= h_tree->getrChild(i)	)
					return false;
		}
	}
	return true;
}

void HTree::setParams(string &fileLine, vector<string> &filePut){	

	int fileLines=0;
	int aParen=0, fParen=0, comma=0;
	int quantElementosFile;
	char c;
	string currElement, aux;
	while (newickf.get(c)) {
		fileLine +=c;
		fileLines++;

	}
	nInsSpc = 0; // inicializar durante a construção

	while (getline (putf,aux)) //enquanto end of file for false continua
    {      
      filePut.push_back(aux);                             
      nInsSpc++;
    }

	quantElementosFile = fileLines; // qnts elementos o arquivo tem	
	// primeira varredura apenas para verificar inconsistencias
	for(int i = 0; i < quantElementosFile; i++){ // faz uma varredura no arquivo
		currElement = fileLine[i];
		if(currElement == "(") aParen++;
		if(currElement == ")") fParen++;
		if(currElement == ",") comma++;
	}
	if(aParen != fParen){

		cout<< "Arquivo inconsistente, parentes não balanceados" <<endl;
		throw;
	}	
	idxInsSpc = aParen +1; // nos folhas
	nNodes = (aParen * 2) + (nInsSpc * 2) +2;
	idxInsAnc = nNodes - aParen -1;	
}


void HTree::parseTree(string fileLine, vector<string> filePut) {

	int posParent = -1;
	string leaf =" ", ancestral =" ", currElement=" ", parent = " "; // salva o atual e o ultimo elemento
	string leftChild=" ", rightChild= " ", comprimeRamoLeft ="", comprimeRamoRight = "";
	int auxiliarNumNos =0, auxiliarGeral =0, auxilarPreencherVetor =0; // usado para fazer as trocas de elementos no vetor
	int indexleftChild =-1, indexrightChild =-1;
	bool alphabeticModeOn = false; 
	
	// regex
	int quantFolhas = idxInsSpc;
	
	smatch m;
  	regex e ("\\([^()]+\\)");
  	regex folhas("\\([A-z0-9_+.#]+|,[A-z0-9_+.#]+"); // achar todas as folhas e separar no vetor
  	regex internos("\\)[A-z0-9_+.#]+|\\)[:;]");

  	// fill empty names
    size_t pos = 0;
    int n_unamed=1;
    while ((pos = fileLine.find("):", pos)) != std::string::npos) {
         fileLine.replace(pos, 2, ")#"+to_string(n_unamed)+":");
         pos += to_string(n_unamed).length()+3;
         n_unamed++;
    }
    // fill root's empty name (if absent)
    for(pos=fileLine.length(); fileLine[pos]!=';'; pos--);
    if(fileLine[--pos]==')')
    	fileLine.replace(pos,2,")#"+to_string(n_unamed)+";");
    // fill new ancestors' names
    for(int i=0; i<getnInsSpc();i++)
    	setName("na#"+to_string(i+1),getIdxInsAnc()-i); //the new ancestors' insertions order is backward oriented
  	cout << "FileLine: " << fileLine << endl << endl;	
	for(int i=0;i<nNodes;i++){		
		setParent(NOPARENT,i);
		setlChild(NOCHILD,i);
		setrChild(NOCHILD,i);
		setBranch(0,i);
		setdRoot(0,i);		
	}
  	// preencher vetor com todas as species
	// usando o regex para pegar todos os quantFolhas	

	string copyNewick = fileLine;
	while (std::regex_search (copyNewick,m,folhas)) {
	    for (int i=0; i<m.size(); ++i) {
	    	auxiliarGeral = m.position(i)+1; // posicão do match (sem o '(' ou ',')
	    	leaf = copyNewick[auxiliarGeral++];	    	
	    	while(copyNewick[auxiliarGeral]!=':')
	    		leaf += copyNewick[auxiliarGeral++];	    	
  		}
		setName(leaf,auxilarPreencherVetor++);
	    copyNewick = m.suffix().str();
  	}
  	// preencher vetor com todas as species
	// usando o regex para pegar todos os nos internos	
	auxilarPreencherVetor = quantFolhas + (nInsSpc * 2) + 1;
	copyNewick = fileLine;
	while (std::regex_search (copyNewick,m,internos)) {
		ancestral = "";
	    for (int i=0; i<m.size(); ++i) {

	    	auxiliarGeral = m.position(i) +1; // posicão do match
		    while(copyNewick[auxiliarGeral]!=':' && copyNewick[auxiliarGeral]!=';') {
		    	ancestral += copyNewick[auxiliarGeral++];	    			
		    }		    
  		} 
  		setName(ancestral,auxilarPreencherVetor);
  		auxilarPreencherVetor++;
	    copyNewick = m.suffix().str();
  	}  	  	 
  	setParent(NOPARENT,nNodes-1); // no raiz não tem um pai
  	
	// logica se da no principio de achar todos os nos folhas pares, em cada loop, dai verificamos o seu devido pai
	// e os "eliminamos" da arvore, criando novos filhos folhas.
	// Para isso, estamos usando a biblioteca Redex, para achar os matchs e fazer o replace em seguida.
	// links: http://www.cplusplus.com/reference/regex/regex_search/
	//		  http://www.cplusplus.com/reference/regex/match_results/position/
	//        http://www.cplusplus.com/reference/regex/regex_replace/
	
	
	//regex logica
	// enquanto tivermos nos para buscar, vamos tirar as folhas
	// sobrara no final apenas o pai raiz
 
	int numTotalNos = nNodes-(2*nInsSpc)-1; 
	while(auxiliarNumNos < numTotalNos -1){	

		leftChild = "";
		rightChild = "";
		comprimeRamoLeft = "";
		comprimeRamoRight = "";
		std::regex_search ( fileLine, m, e );
    	
    	currElement = fileLine[m.position(0)]; // primeiro paranteses dos nos folhas achados    	
    	auxiliarGeral = m.position(0);

    	/* read everything until the ':' charactere is reached */
    	while(fileLine[++auxiliarGeral]!=':')
    		if(fileLine[auxiliarGeral]!=' ') leftChild += fileLine[auxiliarGeral];

   		while(fileLine[++auxiliarGeral]!=',')
    		if(fileLine[auxiliarGeral]!=' ') comprimeRamoLeft += fileLine[auxiliarGeral];

    	while(fileLine[++auxiliarGeral]!=':')
    		if(fileLine[auxiliarGeral]!=' ') rightChild += fileLine[auxiliarGeral];

   		while(fileLine[++auxiliarGeral]!=')')
    		if(fileLine[auxiliarGeral]!=' ') comprimeRamoRight += fileLine[auxiliarGeral];

	    auxiliarGeral++;

    	/* fetch name of the internal node (until ':') or of the root (until ';') */
    	parent="";
    	while(fileLine[auxiliarGeral]!=':' && fileLine[auxiliarGeral]!=';') {
	    	parent += fileLine[auxiliarGeral++];	    			
	    }	 

  		// achar o index entao dos filhos tirados e do pai
    	for(int i=0; i<nNodes; i++){
    		if(name[i] == parent){
    			posParent = i;
    			if( (indexleftChild != -1) and (indexrightChild != -1) ) break; // parar se ja achou indexes
    		}
    		else if(name[i]==rightChild){    			
    			indexrightChild = i;
    			if( (indexleftChild != -1) and (posParent != -1) ) break; 
    		}
    		else if(name[i]==leftChild){    			
    			indexleftChild = i;
    			if( (indexrightChild != -1) and (posParent != -1) ) break;
    		}
    	}
    	// preencher vetores
    	setParent(posParent,indexleftChild);
    	setSide(1,indexleftChild);
    	setParent(posParent,indexrightChild);
    	setSide(0,indexrightChild);
    	setlChild(indexleftChild,posParent);
    	setrChild(indexrightChild,posParent);
    	// comprimento do ramo
    	try{
	    	setBranch(atof(comprimeRamoRight.c_str()),indexrightChild);
	    	setBranch(atof(comprimeRamoLeft.c_str()),indexleftChild);
    	}catch(exception e){

    	}

	  	fileLine = m.prefix().str()+m.suffix().str();

	  	posParent = -1;
	  	// reset variaveis
  		rightChild = "";
  		leftChild = "";
  		comprimeRamoLeft = "";
  		comprimeRamoRight = "";
  		indexrightChild = -1;
  		indexleftChild = -1;
		auxiliarNumNos = auxiliarNumNos + 2; // ou seja, foi retirado 2 filhos
	}
	 // preencher novos put
 	string auxiliarPut[2], auxiliar, put;
  	for (int linePut = 0; linePut < nInsSpc; linePut++)
  	{  	
  		auxiliar = filePut[linePut];
  		put = ""; 
  		auxiliarGeral = 0;
  		alphabeticModeOn = false; 		
	    for (int elemenIndex = 0; elemenIndex < auxiliar.length(); elemenIndex++)
	    {	
	       if (isspace(auxiliar[elemenIndex]) and alphabeticModeOn) 
	       {
	       		auxiliarPut[auxiliarGeral++] = put;	       		
	           	put = "";
	           	alphabeticModeOn = false;
	       }else{
	       		if ( !isspace(auxiliar[elemenIndex]) ){ 
	       			alphabeticModeOn = true;
	        		put += auxiliar[elemenIndex];
	        	}	        	
	       }	 
	    }
	    if(put != ""){
	    	auxiliarPut[auxiliarGeral] = put;
	    }
	    //insert no array especies
	    setName(auxiliarPut[0],quantFolhas+linePut);
	    for (int index = 0; index < nNodes; index++)
	    {
	    	if(name[index] == auxiliarPut[1]){
	    		if(index>=getIdxInsSpc())
	    			setParent(index,quantFolhas+linePut);
	    		else // if the MDCC is a leaf, make its parent become the new MDCC 
	    			setParent(getParent(index),quantFolhas+linePut);
	    		break;
	    	}
	    }	
  	} 
	// Calcular comprimento do ramo ate a raiz
	// usando busca em profundidade
	bool folhaDone = false;
	int visited=0;
	setBranch(0,nNodes-1);	//root has no branch
	setdRoot(0,nNodes-1); 	//root has no distance to himself
	int posRamo = getrChild(nNodes-1);//start with the root's right child;
	while(visited<quantFolhas*2-2){
		// primeiramente, faz uma busca profunda, pela esquerda(mas na vdd tanto faz), e busca um no leaf
		// com isso, sabemos a profundidade de todos os outros folhas, restando então apenas os nos internos
		// essa regra se aplica apenas para arvores filogeneticas
		while(not folhaDone){
			setdRoot(getdRoot(getParent(posRamo))+getBranch(posRamo),posRamo);
			if(getrChild(posRamo) == NOCHILD){ // ou seja, não tem filho(leaf)
				folhaDone = true;
				// temos então o comprimento de todos os folhas da arvore
				// atualizar de todas as folhas então				
				for (int i = 0; i < quantFolhas + nInsSpc; i++)
				{
					setdRoot(getdRoot(posRamo),i);
				}
				visited+= quantFolhas;
				posRamo = getParent(posRamo); // volta entao a posição ramo 1 posição, pois chegou no limite da arvore(leaf)				
				break;
			}
			visited++;
			posRamo = getrChild(posRamo); // proximo filho a direita
		}		
		// fazer a busca em profundidade agr para os nos internos
		// se os dois filhos da raiz, ja tiverem seus comprimentos achados,
		// entao significa q a busca em profundidade foi concluida

		// cheka se elemento atual ainda tem filho 
		if(getrChild(posRamo)!=NOCHILD){
			// se tiver filho da direita e o comp dele ainda n foi calculado
			if(getdRoot(getrChild(posRamo))==0){
				// nova posRamo é entao aquele filho da direita
				visited++;
				posRamo = getrChild(posRamo);
				setdRoot(getdRoot(getParent(posRamo))+getBranch(posRamo),posRamo);
			}
			 // se tiver filho da esquerda e o comp dele ainda n foi calculado
			else if(getdRoot(getlChild(posRamo))==0){
				// nova posRamo é entao aquele filho da direita
				visited++;
				posRamo = getlChild(posRamo);
				setdRoot(getdRoot(getParent(posRamo))+getBranch(posRamo),posRamo);
			}
			// ou seja, aquela sub arvore esta concluida
			else{				
				posRamo = getParent(posRamo); // volta entao a posição ramo 1 posição, pois chegou no limite da arvore(leaf)	
			}	
		}
	}
	//setup insertion sequence
	for(int i=0; i<getnInsSpc();i++)
		setInseq(getIdxInsSpc()+i,i);
}

DTree& HTree::gpuRep(int num_reps) const{	
	size_t rep_size = treeSize * num_reps;
	void *d_replics;
	CHECK(hipMalloc(&d_replics, rep_size));
	hipDeviceProp_t device;
	CHECK(hipGetDeviceProperties(&device,devId));
	int threads = device.warpSize*16;	//TODO: FIGURE OUT WHICH MULTIPLE IS THE BEST
	int blocks = (rep_size/sizeof(int4) + (threads-1)) / threads;
	dim3 grid = dim3(blocks);
	dim3 block = dim3(threads);
	modcpy<<<grid, block>>>(d_replics,devData.getPtr(),rep_size,treeSize);
	CHECK(hipDeviceSynchronize());
	return *new DTree(nNodes,nInsSpc,idxInsSpc,idxInsAnc,num_reps,treeSize,d_replics);
}

__global__ void setup_kernel(long long seed, hiprandState_t* devStates, ushort N){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
    for(i=idx;i<N;i+=gridDim.x*blockDim.x)
    	hiprand_init(seed, i, 0, &devStates[i]);
}

__global__ void insertion(DTree tree, hiprandState_t* devStates){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState state;
	unsigned int i,j,t;
	int taxon, mdcc;
	int ancidx; 	//the put's parent node created to represent the cladogenesis
	int grandpa;
	unsigned int k;

	float depth;	//depth in which the put will be inserted down the subtree rooted at mdcc
	float height;	//height of the tree (distance from leaf to root)
	
	for(k=idx;k<tree.getnTrees();k+=gridDim.x*blockDim.x){	
		tree.setTreeIdx(k);
	    state = devStates[k];
	    height = tree.getdRoot(0); //height of the tree (distance from leaf to root)

	    if (tree.getnInsSpc() > 1) {
		for (i=0; i<tree.getnInsSpc()-1; i++) {
			j = i + hiprand(&state) / (UINT_MAX/(tree.getnInsSpc()-i)+1);
			t = tree.getInseq(j);
			tree.setInseq(tree.getInseq(i),j);
			tree.setInseq(t,i);
			}
	    }
	    float sum;
	    ushort put; //current put going to be inserted
		for(i=0; i<tree.getnInsSpc(); i++){
			t = hiprand(&state);	//path
			put = tree.getInseq(i);
			mdcc = tree.getParent(put);	
			depth = hiprand_uniform(&state) * (height-tree.getdRoot(mdcc));
			taxon = mdcc;
			sum=0;
			do{		
				t>>=1;
				taxon = t&1 ? tree.getlChild(taxon) : tree.getrChild(taxon);
				sum+= tree.getBranch(taxon);			
			}while(sum<depth);
			//after the loop, taxon is the sister clade
			grandpa = tree.getParent(taxon);
			ancidx = tree.getIdxInsAnc()-(put-tree.getIdxInsSpc());	//calculate corresponding ancestor node		
			if(t&1){	//if came from the left
				tree.setrChild(put,ancidx);		//put become the right child
				tree.setlChild(taxon,ancidx);	//the sister clade continue being at left
				tree.setlChild(ancidx,grandpa);//the put's parent node takes place of the sister's clade side
			}			
			else{	//if came from the right
				tree.setlChild(put,ancidx);		//put become the left child
				tree.setrChild(taxon,ancidx);	//the sister clade continue being at right
				tree.setrChild(ancidx,grandpa);//the put's parent node takes place of the sister's clade side
			}
			tree.setParent(grandpa,ancidx);				//set up new ancestor's parent (same of the sister group)
			tree.setSide(t&1,ancidx);									//set up new ancestor's side (same of the sister group)
			tree.setParent(ancidx,put);									//set up PUT's parent
			tree.setSide(!(t&1),put);									//set up PUT's side (the sister's reverse)
			tree.setParent(ancidx,taxon);								//set up sister's new parent
			tree.setBranch(tree.getBranch(taxon)-(sum-depth),ancidx);	//set up new ancestor's branch
			tree.setBranch(sum-depth,taxon);							//set up sister's new branch length
			tree.setBranch(height-(tree.getdRoot(mdcc)+depth),put);		//set up PUT's branch length
			tree.setdRoot (tree.getdRoot(grandpa)+tree.getBranch(ancidx),ancidx);	//set up new ancestor's distance to the root
		}	
	}
}

__host__ __device__ int row_index( int i, int M ){ // retorna o indice da linha
	M--;
    float m = M;
    float row = (-2*m - 1 + sqrt( (4*m*(m+1) - 8*(float)i - 7) )) / -2;
    if( row == (float)(int) row ) row -= 1;
    return (int) row;
}

__host__ __device__ int column_index( int i, int M ){ // retorna o indice da coluna
    int row = row_index( i, M);
    M--;
    return 1 + (i - M * row + row*(row+1) / 2);
}

__global__ void patrix(DTree tree, float* d_matrix){

		tree.setTreeIdx(blockIdx.x);
		uint idx = threadIdx.x;
		ushort row, col, taxon;
		unsigned long long row_bmp, col_bmp; 
		ushort row_len, col_len;
		ushort N = tree.getnNodes();
		ushort nleafs = (N-1)/2;
		uint msize = nleafs*(nleafs-1)/2;

		extern __shared__ ushort s[];

		ushort *parent = s;
		ushort *lchild = parent+N;
		ushort *rchild = lchild+N;

		 uint i;
		for(i=idx;i<N;i+=blockDim.x)
				parent[i] = tree.getParent(i);
		for(i=idx;i<N;i+=blockDim.x)
				lchild[i] = tree.getlChild(i);
		for(i=idx;i<N;i+=blockDim.x)
				rchild[i] = tree.getrChild(i);

		__syncthreads();

		for(i=idx;i<msize;i+=blockDim.x)
		{
			row=row_index(i,nleafs);
			col=column_index(i,nleafs);
			row_bmp=0;
			col_bmp=0;
			row_len=0;
			col_len=0;
			for(taxon=row; parent[taxon]!=NOPARENT; taxon=parent[taxon]){
				row_len++;
				row_bmp<<=1;
				row_bmp|=tree.getSide(taxon);
			}
			for(taxon=col; parent[taxon]!=NOPARENT; taxon=parent[taxon]){
				col_len++;
				col_bmp<<=1;
				col_bmp|=tree.getSide(taxon);
			}
			taxon=tree.getnNodes()-1; 	//start with the root
			if((row_bmp&1)==(col_bmp&1)){	//if the LCA isn't the root

				//printf("\nrow=%d, col=%d\n",row,col);

				//printf("\nrow_bmp=%llu, col_bmp=%llu\n",row_bmp, col_bmp);

				do{
					taxon = row_bmp&1 ? lchild[taxon] : rchild[taxon]; // either row_bmp or col_bmp (same)
					//printf("taxon: %d\n",taxon);
				 	row_bmp>>=1;
				 	col_bmp>>=1;
				 }while((row_bmp&1)==(col_bmp&1));
			}
			d_matrix[blockIdx.x*msize+i] = tree.getdRoot(row)+tree.getdRoot(col)-2*tree.getdRoot(taxon);
	}
}

int main(int argc, char *argv[]){	

	if(argc < 2 || argc >4){
		cout << "Usage: " << argv[0] << " #replications [newick putlist]" << endl;
		exit(EXIT_FAILURE);
	}
	
	long long start_time, end_time;
	struct timeval tv;

	int gpu=0;
	double time_spent;
	int num_reps = atoi(argv[1]);	
	HTree *tree = argc>2 ? new HTree(gpu,argv[2],argv[3]) : new HTree(gpu);
	
	CHECK(hipSetDevice(gpu));
	START_TIMER();
	DTree replics = tree->gpuRep(num_reps);
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to replicate trees: "<<time_spent<<"s\n";	

	cout << "nNodes: " << tree->getnNodes() << endl;
	cout << "nInsSpc: " << tree->getnInsSpc() << endl;
	cout << "idxInsSpc: " << tree->getIdxInsSpc() << endl;
	cout << "idxInsAnc: " << tree->getIdxInsAnc() << endl << endl;
/*	
	if(replics.compareTo(tree))
		cout << "Data does match!" << endl;
	else
		cout << "Data doesn't match" << endl;
*/
	hiprandState_t *devStates;

	hipDeviceProp_t device;
	CHECK(hipGetDeviceProperties(&device,gpu));
	
	int threads = device.warpSize*16; //threads per block; TODO: FIGURE OUT WHICH MULTIPLE IS THE BEST
	int blocks = (num_reps + (threads-1)) / threads;
	dim3 grid(blocks), block(threads);
	
	START_TIMER();
	CHECK(hipMalloc((void**)&devStates, sizeof(hiprandState_t)*num_reps));	
	setup_kernel<<<grid,block>>>(1,devStates,num_reps);
	CHECK(hipDeviceSynchronize());
	insertion<<<grid,block>>>(replics,devStates);	
	CHECK(hipDeviceSynchronize());
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to expand trees: "<<time_spent<<"s\n";	
/*
	replics.print(tree->getNames(),0);
	replics.print(tree->getNames(),1);
	replics.print(tree->getNames(),2);

	START_TIMER();
	float *d_matrix;
	ushort nleafs = (replics.getnNodes()-1)/2;
	ushort msize = nleafs*(nleafs-1)/2;
	CHECK(hipMalloc((void**)&d_matrix, sizeof(float)*msize*num_reps));
	patrix<<<num_reps,256,replics.getnNodes()*(sizeof(ushort)*3)>>>(replics, d_matrix);
	CHECK(hipDeviceSynchronize());
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to generate patrixes: "<<time_spent<<"s\n";	

	//replics.free();

	START_TIMER();
	float *h_matrix = (float*)malloc(sizeof(float)*msize*num_reps);
	CHECK(hipMemcpy(h_matrix, d_matrix, sizeof(float)*msize*num_reps, hipMemcpyDeviceToHost));
	STOP_TIMER(time_spent);
	cout<<"\ntotal time spent to copy patrixes to CPU: "<<time_spent<<"s\n";	
*/
	CHECK(hipDeviceReset());
	exit(EXIT_SUCCESS);	
}
